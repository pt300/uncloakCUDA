#include "hip/hip_runtime.h"
#include <string.h>
#include <stdbool.h>

#include "hash.h"
#include "ip6.h"

#include "cliondoesnthandlecuda.h"

char htable[] = "def-----012345678abc"; //17  a==9
__device__ char dtable[] = "def-----012345678abc";

bool is_ip6(char *host) {
	int chars, doots;

	doots = 0;
	chars = 4;
	while(*host != '\0') {
		if(*host == ':') {
			chars = 4;
			doots++;
		}
		else if((*host >= 'a' && *host <= 'z') ||
				(*host >= '0' && *host <= '9')) {
			if(chars-- == 0) {
				return false;
			}
		}
		else {
			return false;
		}
		host++;
	}

	if(doots < 2) {
		return false;
	}

	return true;
}

size_t find_beginning_ip6(char *host) {
	size_t where;

	for(where = 0; host[where] != '\0' && host[where] < 'g'; where++);

	return where;
}

void ip6_from_hash(uint32_t hash, uint32_t disamb, char *host, size_t begins) {
	size_t i;
	int temp;

	for(i = begins; host[i] != '\0'; i++) {
		if(host[i] != ':') {
			temp = host[i] - 'g';
			temp -= hash % 20;

			if(temp < 0) {
				temp += 20;
			}

			if(temp == 17) {
				host[i] = disamb & 1 ? '9' : 'a';
				disamb >>= 1;
			}
			else {
				host[i] = htable[temp];
			}
			ROTATE32_LEFT(hash);
		}
	}
}

__global__ void
test_hash_ip6(uint32_t shift, unsigned int *ret_n, uint32_t *ret_arr, size_t begins, uint32_t start_hash, char *host) {
	uint32_t hash_tested, hash_string;
	uint32_t hash, disamb_max, disamb, cursor;
	size_t i;
	int temp;


	hash = hash_tested = shift + blockIdx.x * blockDim.x + threadIdx.x;
	hash_string = start_hash;
	disamb = 0;
	disamb_max = ~disamb;

	for(i = begins; host[i] != '\0'; i++) {
		if(host[i] != ':') {
			temp = host[i] - 'g';
			temp -= hash % 20;

			if(temp < 0) {
				temp += 20;
			}

			if(temp < 8 && temp > 2) {
				return;
			}

			if(temp == 17) {
				disamb_max <<= 1;
			}

			hash_string = fnv_hash_streamone(hash_string, dtable[temp]);
			ROTATE32_LEFT(hash);
		}
		else {
			hash_string = fnv_hash_streamone(hash_string, ':');
		}
	}

	if(hash_string == hash_tested) {
		atomicMin(ret_n, 128 * 2);
		i = atomicAdd(ret_n, 1);
		atomicMin(ret_n, 128 * 2);
		ret_arr[i * 2 + 0] = hash_string;
		ret_arr[i * 2 + 1] = disamb;
	}

	disamb_max = ~disamb_max;

	while(disamb++ != disamb_max) {
		for(i = begins, cursor = 1,
			hash = hash_tested, hash_string = start_hash; host[i] != '\0'; i++) {
			if(host[i] != ':') {
				temp = host[i] - 'g';
				temp -= hash % 20;

				if(temp < 0) {
					temp += 20;
				}

				if(temp == 17) {
					hash_string = fnv_hash_streamone(hash_string, (char) (disamb & cursor ? '9' : 'a'));
					cursor <<= 1;
				}
				else {
					hash_string = fnv_hash_streamone(hash_string, dtable[temp]);
				}
				ROTATE32_LEFT(hash);
			}
			else {
				hash_string = fnv_hash_streamone(hash_string, ':');
			}
		}

		if(hash_string == hash_tested) {
			atomicMin(ret_n, 128 * 2);
			i = atomicAdd(ret_n, 1);
			atomicMin(ret_n, 128 * 2);
			ret_arr[i * 2 + 0] = hash_string;
			ret_arr[i * 2 + 1] = disamb;
		}

	}

}