#include "hip/hip_runtime.h"
#include <string.h>
#include <ctype.h>

#include "hash.h" //this is the only way that makes it work?????
#include "host.h"

/*
 * 2 copies, one used to retrieve string from hash on host
 * other is for computations on device
 */

__device__ static const char *dtable_letters = "hijklmnopqrstuvwxyzabcdefg";
__device__ static const char *dtable_digits = "2345678901";
static const char *table_letters = "hijklmnopqrstuvwxyzabcdefg";
static const char *table_digits = "2345678901";

__global__ void test_hash_host(uint32_t shift, unsigned int *ret_n, uint32_t *ret_arr, size_t letters, char *host) {
	uint32_t hash_tested, hash_string;
	uint32_t hash_digits, hash_letters;
	size_t i;
	int temp;


	hash_digits = hash_letters = hash_tested = shift + blockIdx.x * blockDim.x + threadIdx.x;
	hash_string = FNV1_32_INIT;
	ROTATE32_LEFTN(hash_digits, letters);

	for(i = 0; host[i] != '\0' && host[i] != '.'; i++) {
		if(host[i] >= 'a' && host[i] <= 'z') {
			temp = host[i] - 'a';
			temp -= hash_letters % 26;
			if(temp < 0) {
				temp += 26;
			}
			hash_string = fnv_hash_streamone(hash_string, dtable_letters[temp]);
			ROTATE32_LEFT(hash_letters);
		}
		else if(host[i] >= '0' && host[i] <= '9') {
			temp = host[i] - '0';
			temp -= hash_digits % 10;
			if(temp < 0) {
				temp += 10;
			}
			hash_string = fnv_hash_streamone(hash_string, dtable_digits[temp]);
		}
		ROTATE32_LEFT(hash_digits);
	}
	if(host[i] == '.') {
		hash_string = fnv_hash_streamend(hash_string, host + i);
	}

	if(hash_string == hash_tested) {
		atomicMin(ret_n, 128);
		ret_arr[atomicAdd(ret_n, 1)] = hash_string;
	}
}

bool is_valid_host(char *host) {
	for(; *host != '\0'; host++) {
		if(!(isdigit(*host) || islower(*host) || *host == '-' || *host == '.')) {
			return false;
		}
	}

	return true;
}

size_t count_letters(char *host) {
	size_t chars;

	for(chars = 0; *host != '\0' && *host != '.'; host++) {
		if(islower(*host)) {
			chars++;
		}
	}

	return chars;
}

void host_from_hash(uint32_t hash, char *host) {
	size_t i;
	int temp;

	for(i = 0; host[i] != '\0' && host[i] != '.'; i++) {
		if(islower(host[i])) {
			temp = host[i] - 'a';
			temp -= hash % 26;
			if(temp < 0) {
				temp += 26;
			}
			//we just assume it uses 64bit values during calculations
			host[i] = table_letters[temp];
			ROTATE32_LEFT(hash);
		}
	}

	for(i = 0; host[i] != '\0' && host[i] != '.'; i++) {
		if(isdigit(host[i])) {
			temp = host[i] - '0';
			temp -= hash % 10;
			if(temp < 0) {
				temp += 10;
			}
			host[i] = table_digits[temp];
		}
		ROTATE32_LEFT(hash);
	}
}