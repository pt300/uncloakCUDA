#include "hip/hip_runtime.h"
#include <string.h>
#include <stdbool.h>
#include <stdio.h>
#include <alloca.h>

#include "hash.h"
#include "ip4.h"

#include "cliondoesnthandlecuda.h"

bool is_ip4(char *host) {
	int dots;

	dots = 0;
	while(*host != '\0') {
		if(*host == '.') {
			if(dots++ == 3) {
				return false;
			}
		}
		else if((*host >= 'g' && *host <= 'z' && dots < 2) ||
				(*host >= '0' && *host <= '9' && dots > 1)) {
			return false;
		}


		host++;
	}

	return true;
}

size_t find_beginning_ip4(char *host) {
	size_t where;
	int doot;

	for(where = 0, doot = 1; host[where] != '\0'; where++) {
		if(host[where] == '.') {
			if(doot == 1) {
				doot = 0;
			}
			else {
				return where + 1;
			}
		}
	}

	return 0;
}

void ip4_from_hash(uint32_t hash, char *host, size_t begins) {
	size_t i;
	int temp;

	for(i = begins; host[i] != '\0'; i++) {
		if(host[i] != '.') {
			temp = host[i] - 'g';
			temp -= hash % 20;
			if(temp < 0) {
				temp += 20;
			}
			temp -= 8;
			host[i] = (char) (temp + '0');
			ROTATE32_LEFT(hash);
		}
	}
}

__global__ void test_hash_ip4(uint32_t shift, unsigned int *ret_n, uint32_t *ret_arr, size_t begins,
							  uint32_t start_hash, char *host) {
	uint32_t hash_tested, hash_string;
	uint32_t hash;
	size_t i;
	int temp;


	hash = hash_tested = shift + blockIdx.x * blockDim.x + threadIdx.x;
	hash_string = start_hash;

	for(i = begins; host[i] != '\0'; i++) {
		if(host[i] != '.') {
			temp = host[i] - 'g';
			temp -= hash % 20;
			if(temp < 0) {
				temp += 20;
			}
			if(temp < 8 || temp > 17) {
				return;
			}
			//TODO: add check?
			//should be in range <8; 17>
			temp -= 8;
			hash_string = fnv_hash_streamone(hash_string, (char) (temp + '0'));
			ROTATE32_LEFT(hash);
		}
		else {
			hash_string = fnv_hash_streamone(hash_string, '.');
		}
	}

	if(hash_string == hash_tested) {
		atomicMin(ret_n, 128);
		ret_arr[atomicAdd(ret_n, 1)] = hash_string;
	}
}

int size_to_threads(int s) {
	return s == 3 ? 156 : s == 2 ? 89 : 10;
}

int size_to_shift(int s) {
	return s == 3 ? 100 : s == 2 ? 10 : 0;
}

int get_n_size(char *host, int n) {
	int sep, num;

	for(sep = num = 0; *host != '\0' && sep != n; host++) {

		if(*host == '.') {
			sep++;
		}
		else if(sep == n - 1) {
			num++;
		}
	}

	return num;
}

__global__ void test_str_ip4(unsigned int *ret_n, uint32_t *ret_arr, uint32_t start_hash, size_t begins,
							 int n3_shift, int n4_shift, char *host) {
	char fuckingarray[8], *ch;
	uint32_t hash;
	uint32_t n3, n4;
	size_t i;

	ch = fuckingarray;
	//sprintf(ch, "%hhu.%hhu", n3_shift + blockIdx.x, n4_shift + threadIdx.x);
	n3 = n3_shift + blockIdx.x;
	n4 = n4_shift + threadIdx.x;

	/*
	 * the evil part
	 */

	if(n3 >= 100) {
		*ch++ = (char) (n3 / 100 + '0');
		n3 %= 100;
		*ch++ = (char) (n3 / 10 + '0');
		n3 %= 10;
	}
	else if(n3 >= 10) {
		*ch++ = (char) (n3 / 10 + '0');
		n3 %= 10;
	}
	*ch++ = (char) (n3 + '0');
	*ch++ = '.';
	if(n4 >= 100) {
		*ch++ = (char) (n4 / 100 + '0');
		n4 %= 100;
		*ch++ = (char) (n4 / 10 + '0');
		n4 %= 10;
	}
	else if(n4 >= 10) {
		*ch++ = (char) (n4 / 10 + '0');
		n4 %= 10;
	}
	*ch++ = (char) (n4 + '0');
	*ch = '\0';

	n3 = (uint8_t) (n3_shift + blockIdx.x);
	n4 = (uint8_t) (n4_shift + threadIdx.x);


	hash = start_hash;
	hash = fnv_hash_streamend(hash, fuckingarray);

	for(i = begins, ch = fuckingarray; *ch != '\0'; i++, ch++) {
		if(host[i] != '.') {
			if((hash + *ch) % 20 != host[i] - 'g') {
				return;
			}
			ROTATE32_LEFT(hash);
		}
	}


	atomicMin(ret_n, 128);
	ret_arr[atomicAdd(ret_n, 1)] = (n3 << 8) | n4;
}