#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>

#include "cliondoesnthandlecuda.h"

#define HOSTLEN 63

//the lazy way
#include "host.h"
#include "ip4.h"
#include "ip6.h"
#include "hash.h"

enum type {
	HOSTNAME,
	IP4,
	IP6
};

unsigned int handle_host(char *host, size_t len);
unsigned int handle_ip4(char *host, size_t len);
unsigned int handle_ip4_bf(char *host, size_t len);
unsigned int handle_ip6(char *host, size_t len);

int main(int argc, char **argv) {
	size_t len;
	char *host;
	unsigned int found;
	enum type wearedealingwith;

	int dev;
	struct hipDeviceProp_t prop;

	/*
	 * Host/IP preparation stuff
	 */

	if(argc < 2) {
		fprintf(stderr, "Gib me a cloaked hostname or IP number. At most %i chars long\n", HOSTLEN);
		return EXIT_FAILURE;
	}

	if(strlen(argv[1]) > HOSTLEN || !argv[1][0]) {
		fprintf(stderr, "Size %li is fooken illegal.\n", strlen(argv[1]));
		return EXIT_FAILURE;
	}

	len = strlen(argv[1]);
	host = (char *) alloca(len + 1); //that pointer cast, fuck you nvidia
	strcpy(host, argv[1]);

	if(is_ip4(host)) {
		wearedealingwith = IP4;
		printf("IPv4 address detected\n");
	}
	else if(is_ip6(host)) {
		wearedealingwith = IP6;
		printf("IPv6 address detected\n");
	}
	else if(is_valid_host(host)) {
		wearedealingwith = HOSTNAME;
		printf("Regular host name detected\n");
	}
	else {
		printf("I have no idea what to do with this crap you gave me\n");
		return EXIT_FAILURE;
	}

	/*
	 * CUDA init stuff
	 */

	if(hipGetDeviceCount(&dev) != hipSuccess) {
		fprintf(stderr, "CUDA seems to be broken for you.\n"
				"One way to fix it is running _ONCE_ any program that uses CUDA as root (sudo).\n"
				"If that doesn't work for you, fix your drivers.\n");
		return EXIT_FAILURE;
	}

	printf("Available devices count: %i\n", dev);
	if(dev == 0) {
		puts("No devices found.\n");
		return EXIT_FAILURE;
	}
	while(dev--) {
		hipGetDeviceProperties(&prop, dev);
		if(prop.name[0] == 'G' &&
		   prop.name[1] == 'e')
			printf("\tGay%s\n", prop.name + 2);
		else
			printf("\t%s\n", prop.name);
	}
	puts("");


	/*
	 * THE STUFF
	 */

	switch(wearedealingwith) {
		case HOSTNAME:
			found = handle_host(host, len);
			break;
		case IP4:
			found = handle_ip4_bf(host, len);
			break;
		case IP6:
			found = handle_ip6(host, len);
			break;
	}

	printf("\rFound %i matches. Exiting...\n", found);

	return EXIT_SUCCESS;
}

unsigned int handle_host(char *host, size_t len) {
	uint64_t done;
	uint32_t *dmatch_array, hhash;
	unsigned int *dmatches, matches, hmatches;
	char *dhost, *phost;
	size_t letters;
	int wrote;

	phost = (char *) alloca(len + 1);

	hipMalloc((void **) &dmatches, sizeof *dmatches);
	hipMalloc((void **) &dmatch_array, 128 * sizeof *dmatch_array);
	hipMalloc((void **) &dhost, len + 1); //don't forget about null you dumbfuck

	hipMemset(dmatches, 0, sizeof *dmatches);
	hipMemcpy(dhost, host, len + 1, hipMemcpyHostToDevice);

	done = 0;
	matches = 0;


	letters = count_letters(host);

	do {
		test_hash_host<<<1024, 1024>>>(done, dmatches, dmatch_array, letters, dhost);
		done += 1024 * 1024;

		hipMemcpy((void *) &hmatches, dmatches, sizeof *dmatches, hipMemcpyDeviceToHost);
		matches += hmatches;
		hipMemset(dmatches, 0, sizeof *dmatches);
		while(hmatches--) {
			hipMemcpy(&hhash, dmatch_array + hmatches * sizeof *dmatch_array, sizeof *dmatch_array,
					   hipMemcpyDeviceToHost);
			printf("\r%*c\r", wrote, ' ');
			strcpy(phost, host);
			host_from_hash(hhash, phost);
			puts(phost);
		}
		wrote = printf("\r%f%%, %u match", (float) done * 100 / ((float) UINT32_MAX + 1), matches);
		fflush(stdout);
	} while(done < (uint64_t) UINT32_MAX + 1);

	printf("\r%*c\r", wrote, ' ');

	hipFree(dmatches);
	hipFree(dmatch_array);
	hipFree(dhost);

	return matches;
}

unsigned int handle_ip4(char *host, size_t len) {
	uint64_t done;
	uint32_t *dmatch_array, hhash, start_hash;
	unsigned int *dmatches, matches, hmatches;
	char *dhost, *phost;
	size_t beg;
	int wrote;

	phost = (char *) alloca(len + 1);

	hipMalloc((void **) &dmatches, sizeof *dmatches);
	hipMalloc((void **) &dmatch_array, 128 * sizeof *dmatch_array);
	hipMalloc((void **) &dhost, len + 1);

	hipMemset(dmatches, 0, sizeof *dmatches);
	hipMemcpy(dhost, host, len + 1, hipMemcpyHostToDevice);

	done = 0;
	matches = 0;
	wrote = 0;

	beg = find_beginning_ip4(host);
	start_hash = fnv_hash_n(host, beg);

	do {
		test_hash_ip4<<<1024, 1024>>>(done, dmatches, dmatch_array, beg, start_hash, dhost);
		done += 1024 * 1024;

		hipMemcpy((void *) &hmatches, dmatches, sizeof *dmatches, hipMemcpyDeviceToHost);
		matches += hmatches;
		hipMemset(dmatches, 0, sizeof *dmatches);
		while(hmatches--) {
			hipMemcpy(&hhash, dmatch_array + hmatches * sizeof *dmatch_array, sizeof *dmatch_array,
					   hipMemcpyDeviceToHost);
			printf("\r%*c\r", wrote, ' ');
			strcpy(phost, host);
			ip4_from_hash(hhash, phost, beg);
			puts(phost);
		}
		wrote = printf("\r%f%%, %u match", (float) done * 100 / ((float) UINT32_MAX + 1), matches);
		fflush(stdout);
	} while(done < (uint64_t) UINT32_MAX + 1);

	printf("\r%*c\r", wrote, ' ');

	hipFree(dmatches);
	hipFree(dmatch_array);
	hipFree(dhost);

	return matches;
}

unsigned int handle_ip4_bf(char *host, size_t len) {
	uint32_t *dmatch_array, hhash, start_hash;
	unsigned int *dmatches, matches, hmatches;
	char *dhost;
	size_t beg;
	int n3, n4, blocks, threads, shift_n3, shift_n4;

	hipMalloc((void **) &dmatches, sizeof *dmatches);
	hipMalloc((void **) &dmatch_array, 128 * sizeof *dmatch_array);
	hipMalloc((void **) &dhost, len + 1);

	hipMemset(dmatches, 0, sizeof *dmatches);
	hipMemcpy(dhost, host, len + 1, hipMemcpyHostToDevice);

	matches = 0;

	beg = find_beginning_ip4(host);
	start_hash = fnv_hash_n(host, beg);

	n3 = get_n_size(host, 3);
	n4 = get_n_size(host, 4);

	blocks = size_to_threads(n3);
	threads = size_to_threads(n4);
	shift_n3 = size_to_shift(n3);
	shift_n4 = size_to_shift(n4);

	test_str_ip4<<<blocks, threads>>>(dmatches, dmatch_array, start_hash, beg, shift_n3, shift_n4, dhost);

	hipMemcpy((void *) &hmatches, dmatches, sizeof *dmatches, hipMemcpyDeviceToHost);
	matches += hmatches;
	hipMemset(dmatches, 0, sizeof *dmatches);
	while(hmatches--) {
		hipMemcpy(&hhash, dmatch_array + hmatches * sizeof *dmatch_array, sizeof *dmatch_array,
				   hipMemcpyDeviceToHost);
		printf("%.*s%hu.%hu\n", (int) beg, host, (hhash >> 8) & 0xFF, hhash & 0xFF);
	}

	hipFree(dmatches);
	hipFree(dmatch_array);
	hipFree(dhost);

	return matches;
}

#pragma clang diagnostic push
#pragma ide diagnostic ignored "OCDFAInspection"

unsigned int handle_ip6(char *host, size_t len) {
	uint64_t done;
	uint32_t *dmatch_array, hhash, disamb, start_hash;
	unsigned int *dmatches, matches, hmatches;
	char *dhost, *phost;
	size_t beg;
	int wrote;

	phost = (char *) alloca(len + 1);

	hipMalloc((void **) &dmatches, sizeof *dmatches);
	hipMalloc((void **) &dmatch_array, 128 * 2 * sizeof *dmatch_array);
	hipMalloc((void **) &dhost, len + 1);

	hipMemset(dmatches, 0, sizeof *dmatches);
	hipMemcpy(dhost, host, len + 1, hipMemcpyHostToDevice);

	done = 0;
	matches = 0;
	wrote = 0;

	beg = find_beginning_ip6(host);
	start_hash = fnv_hash_n(host, beg);

	do {
		test_hash_ip6<<<1024, 1024>>>(done, dmatches, dmatch_array, beg, start_hash, dhost);
		done += 1024 * 1024;

		hipMemcpy((void *) &hmatches, dmatches, sizeof *dmatches, hipMemcpyDeviceToHost);
		matches += hmatches;
		hipMemset(dmatches, 0, sizeof *dmatches);
		while(hmatches--) {
			hipMemcpy(&hhash, dmatch_array + hmatches * sizeof *dmatch_array * 2, sizeof *dmatch_array,
					   hipMemcpyDeviceToHost);
			hipMemcpy(&disamb, dmatch_array + hmatches * sizeof *dmatch_array * 2 + 1, sizeof *dmatch_array,
					   hipMemcpyDeviceToHost);
			printf("\r%*c\r", wrote, ' ');
			strcpy(phost, host);
			ip6_from_hash(hhash, disamb, phost, beg);
			puts(phost);
		}
		wrote = printf("\r%f%%, %u match", (float) done * 100 / ((float) UINT32_MAX + 1), matches);
		fflush(stdout);
	} while(done < (uint64_t) UINT32_MAX + 1);

	printf("\r%*c\r", wrote, ' ');

	hipFree(dmatches);
	hipFree(dmatch_array);
	hipFree(dhost);

	return matches;
}

#pragma clang diagnostic pop